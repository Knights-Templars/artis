#include "hip/hip_runtime.h"
// #include <math.h>
// #include <stdint.h>
// #include <stdio.h>
// #include "hip/hip_runtime.h"
#include "radfield.h"
#include "artisoptions.h"

__global__ void kernel_radfield(double nu, struct radfieldbin *radfieldbins_thiscell, double *radfieldbin_nu_upper, double *radfieldjnu)
{
    const int binindex = threadIdx.x + blockIdx.x * blockDim.x;
    const float bin_T_R = radfieldbins_thiscell[binindex].T_R;
    const float bin_W = radfieldbins_thiscell[binindex].W;
    const double bin_nu_lower = binindex == 0 ? nu_lower_first_initial : radfieldbin_nu_upper[binindex - 1];
    const double bin_nu_upper = radfieldbin_nu_upper[binindex];
    if (bin_nu_upper > nu && bin_nu_lower <= nu)
    {
        // printf("CUDAkernel: nu %lg binindex %d nu_lower %lg nu_upper %lg T_R %g W %g\n", nu, binindex, bin_nu_lower, bin_nu_upper, bin_T_R, bin_W);
        *radfieldjnu = bin_W * TWOHOVERCLIGHTSQUARED * pow(nu, 3) / expm1(HOVERKB * nu / bin_T_R);
        // printf("    radfieldjnu %g\n", *radfieldjnu);
    }
}


__device__ double photoionization_crosssection_fromtable(float *photoion_xs, double nu_edge, double nu, int NPHIXSPOINTS, double NPHIXSNUINCREMENT)
/// Calculates the photoionisation cross-section at frequency nu out of the atomic data.
/// Input: - edge frequency nu_edge of the desired bf-continuum
///        - nu
{
  float sigma_bf;
  const double ireal = (nu / nu_edge - 1.0) / NPHIXSNUINCREMENT;
  const int i = floor(ireal);

  if (i < 0)
  {
    sigma_bf = 0.0;
  }
  else if (i < NPHIXSPOINTS - 1)
  {
    // sigma_bf = elements[element].ions[ion].levels[level].photoion_xs[i];

    const double sigma_bf_a = photoion_xs[i];
    const double sigma_bf_b = photoion_xs[i + 1];
    const double factor_b = ireal - i;
    sigma_bf = ((1. - factor_b) * sigma_bf_a) + (factor_b * sigma_bf_b);
  }
  else
  {
    const double last_phixs_nuovernuedge = (1.0 + NPHIXSNUINCREMENT * (NPHIXSPOINTS - 1));
    const double nu_max_phixs = nu_edge * last_phixs_nuovernuedge; //nu of the uppermost point in the phixs table
    sigma_bf = photoion_xs[NPHIXSPOINTS-1] * pow(nu_max_phixs / nu, 3);
  }

  return sigma_bf;
}


// const int blocksize = 10;

__global__ void kernel_corrphotoion_integral(
  struct radfieldbin *radfieldbins_thiscell, double *radfieldbin_nu_upper, double nu_edge, float *photoion_xs,
  double departure_ratio, float T_e, double *integral, int NPHIXSPOINTS, double NPHIXSNUINCREMENT)
/// Integrand to calculate the rate coefficient for photoionization
/// using gsl integrators. Corrected for stimulated recombination.
{
  if (threadIdx.x >= RADFIELDBINCOUNT)
    return;

  __shared__ double part_integral[RADFIELDBINCOUNT];

  const int binindex = threadIdx.x;

  const float bin_T_R = radfieldbins_thiscell[binindex].T_R;
  const float bin_W = radfieldbins_thiscell[binindex].W;
  const double bin_nu_lower = binindex == 0 ? nu_lower_first_initial : radfieldbin_nu_upper[binindex - 1];
  const double bin_nu_upper = radfieldbin_nu_upper[binindex];

  const double delta_nu = (bin_nu_upper - bin_nu_lower);

  // const int binpiece = blockIdx.x;
  const int binpiece = 0;

  const double nu = bin_nu_lower + binpiece * delta_nu;

  #if (SEPARATE_STIMRECOMB)
    const double corrfactor = 1.0;
  #else
    double corrfactor = 1. - departure_ratio * exp(-HOVERKB * nu / T_e);
    if (corrfactor < 0)
      corrfactor = 0.;
  #endif

  // printf("kernel_corrphotoion_integral: nu %lg binindex %d nu_lower %lg nu_upper %lg T_R %g W %g\n", nu, binindex, bin_nu_lower, bin_nu_upper, bin_T_R, bin_W);

  const double Jnu = bin_W * TWOHOVERCLIGHTSQUARED * pow(nu, 3) / expm1(HOVERKB * nu / bin_T_R);

  const float sigma_bf = photoionization_crosssection_fromtable(photoion_xs, nu_edge, nu, NPHIXSPOINTS, NPHIXSNUINCREMENT);

  part_integral[threadIdx.x] = ONEOVERH * sigma_bf / nu * Jnu * corrfactor * delta_nu;

  // printf("kernel_corrphotoion_integral sigma_bf %g T_e %g corrfactor %lg part_integral %g\n", sigma_bf, T_e, corrfactor, part_integral);

  __syncthreads();

  // const int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadIdx.x == 0)
  {
    *integral = 0;
    for (unsigned int s = 0; s < RADFIELDBINCOUNT; s++) // change to blockDim.x
    {
      *integral = *integral + part_integral[s];
    }
  }

  __syncthreads();
}


double radfield_gpu(double nu, int modelgridindex)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed. CUDA-capable GPU installed?");
        abort();
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsPerBlock(RADFIELDBINCOUNT, 1, 1);
    dim3 numBlocks(1, 1, 1);

    double *radfieldjnu;

    hipMallocManaged(&radfieldjnu, sizeof(double));
    *radfieldjnu = 0;

    kernel_radfield<<<numBlocks, threadsPerBlock>>>(nu, radfieldbins[modelgridindex], radfieldbin_nu_upper, radfieldjnu);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        abort();
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        abort();
    }

    double result = *radfieldjnu;
    hipFree(radfieldjnu);
    return result;
}


double calculate_corrphotoioncoeff_integral_gpu(int modelgridindex, double nu_edge, float *photoion_xs, double departure_ratio, float T_e)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed. CUDA-capable GPU installed?");
        abort();
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsPerBlock(RADFIELDBINCOUNT, 1, 1);
    dim3 numBlocks(1, 1, 1);

    double *integral;

    hipMallocManaged(&integral, sizeof(double));
    *integral = 0;

    cudaStatus = hipDeviceSynchronize();

    kernel_corrphotoion_integral<<<numBlocks, threadsPerBlock>>>(
      radfieldbins[modelgridindex], radfieldbin_nu_upper, nu_edge, photoion_xs, departure_ratio, T_e, integral, NPHIXSPOINTS, NPHIXSNUINCREMENT);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        abort();
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        abort();
    }

    double result = *integral;
    hipFree(integral);
    return result;
}

