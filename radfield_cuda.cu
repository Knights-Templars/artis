#include "hip/hip_runtime.h"
#include <assert.h>
#include "artisoptions.h"
#include "atomic.h"
#include "atomic_cuda.cuh"
#include "radfield.h"

extern double *dev_radfieldbin_nu_upper;
extern struct radfieldbin **dev_radfieldbins;


__global__ void kernel_radfield(double nu, struct radfieldbin *radfieldbins_thiscell, double *radfieldbin_nu_upper, double *radfieldjnu)
{
    const int binindex = threadIdx.x + blockIdx.x * blockDim.x;
    const float bin_T_R = radfieldbins_thiscell[binindex].T_R;
    const float bin_W = radfieldbins_thiscell[binindex].W;
    const double bin_nu_lower = binindex == 0 ? nu_lower_first_initial : radfieldbin_nu_upper[binindex - 1];
    const double bin_nu_upper = radfieldbin_nu_upper[binindex];
    if (bin_nu_lower <= nu && bin_nu_upper > nu)
    {
      // printf("CUDAkernel: nu %lg binindex %d nu_lower %lg nu_upper %lg T_R %g W %g\n", nu, binindex, bin_nu_lower, bin_nu_upper, bin_T_R, bin_W);
      *radfieldjnu = bin_W * TWOHOVERCLIGHTSQUARED * pow(nu, 3) / expm1(HOVERKB * nu / bin_T_R);
      // printf("    radfieldjnu %g\n", *radfieldjnu);
    }
}


__device__ int select_bin_gpu(double nu, double *radfieldbin_nu_upper)
{
  // linear search one by one until found
  if (nu >= radfieldbin_nu_upper[RADFIELDBINCOUNT - 1])
    return -1; // out of range, nu higher than highest bin
  else if (nu < nu_lower_first_initial)
    return -2; // out of range, nu lower than lowest bin
  else
  {
    for (int binindex = 0; binindex < RADFIELDBINCOUNT; binindex++)
    {
      if (radfieldbin_nu_upper[binindex] > nu)
      {
        return binindex;
      }
    }

    return -3;
  }
}

const int integralsamplesperxspoint = 8; // must be an even number for Simpsons rule to work

__global__ void kernel_corrphotoion_integral(
  struct radfieldbin *radfieldbins_thiscell, double *radfieldbin_nu_upper, double nu_edge, float *photoion_xs,
  double departure_ratio, float T_e, double *integral, int NPHIXSPOINTS, double NPHIXSNUINCREMENT)
/// Integrand to calculate the rate coefficient for photoionization
/// using gsl integrators. Corrected for stimulated recombination.
{
  extern __shared__ double part_integral[];
  // __shared__ double part_integral[integralsamplesperxspoint * 100];


  if (threadIdx.x < integralsamplesperxspoint && threadIdx.y < NPHIXSPOINTS)
  {
    // const double last_phixs_nuovernuedge = (1.0 + NPHIXSNUINCREMENT * (NPHIXSPOINTS - 1));

    const double nu = nu_edge * (1. + (NPHIXSNUINCREMENT * (threadIdx.y + (threadIdx.x / integralsamplesperxspoint))));

    const int sampleindex = threadIdx.y * integralsamplesperxspoint + threadIdx.x;

    const int binindex = select_bin_gpu(nu, radfieldbin_nu_upper);

    if (binindex < 0)
    {
      part_integral[sampleindex] = 0.;
    }
    else
    {
      const float bin_T_R = radfieldbins_thiscell[binindex].T_R;
      const float bin_W = radfieldbins_thiscell[binindex].W;
      // const double bin_nu_lower = binindex == 0 ? nu_lower_first_initial : radfieldbin_nu_upper[binindex - 1];
      // const double bin_nu_upper = radfieldbin_nu_upper[binindex];

      const double Jnu = bin_W * TWOHOVERCLIGHTSQUARED * pow(nu, 3) / expm1(HOVERKB * nu / bin_T_R);

      const double delta_nu = nu_edge * (NPHIXSNUINCREMENT / integralsamplesperxspoint);

      #if (SEPARATE_STIMRECOMB)
        const double corrfactor = 1.0;
      #else
        double corrfactor = 1. - departure_ratio * exp(-HOVERKB * nu / T_e);
        if (corrfactor < 0)
          corrfactor = 0.;
      #endif

      const float sigma_bf = photoionization_crosssection_fromtable_gpu(photoion_xs, nu_edge, nu, NPHIXSPOINTS, NPHIXSNUINCREMENT);

      const int lastsampleindex = (NPHIXSPOINTS - 1) * integralsamplesperxspoint + (integralsamplesperxspoint - 1);

      // Simpson's rule integral (will later be divided by 3)
      // n must be odd
      // integral = (xn - x0) / 3 * {f(x_0) + 4 * f(x_1) + 2 * f(x_2) + ... + 4 * f(x_1) + f(x_n-1)}
      // weights e.g., 1,4,2,4,2,4,1
      double weight = 0.;
      if (sampleindex == 0 || sampleindex == lastsampleindex)
      {
        weight = 1.;
      }
      else if (sampleindex % 2 == 0)
      {
        weight = 2.;
      }
      else
      {
        weight = 4.;
      }

      part_integral[sampleindex] = weight * ONEOVERH * sigma_bf / nu * Jnu * corrfactor * delta_nu;
    }
  }

  __syncthreads();

  if (threadIdx.x == 0)
  {
    for (unsigned int x = 1; x < integralsamplesperxspoint; x++)
    {
      const int firstsampleindex = threadIdx.y * integralsamplesperxspoint;
      part_integral[firstsampleindex] += part_integral[firstsampleindex + x];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0)
  {
    double total = 0.;
    for (unsigned int y = 0; y < NPHIXSPOINTS; y++)
    {
      total += part_integral[y * integralsamplesperxspoint];
    }
    *integral = total / 3.;
  }

  __syncthreads();
}


double calculate_corrphotoioncoeff_integral_gpu(int modelgridindex, double nu_edge, float *photoion_xs, double departure_ratio, float T_e)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed. CUDA-capable GPU installed?");
        abort();
    }

    void *dev_integral;

    cudaStatus = hipMalloc(&dev_integral, sizeof(double));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == hipSuccess);

    dim3 threadsPerBlock(integralsamplesperxspoint, NPHIXSPOINTS, 1);
    dim3 numBlocks(1, 1, 1);
    size_t sharedsize = sizeof(double) * NPHIXSPOINTS * integralsamplesperxspoint;

    kernel_corrphotoion_integral<<<numBlocks, threadsPerBlock, sharedsize>>>(
      dev_radfieldbins[modelgridindex], dev_radfieldbin_nu_upper, nu_edge, photoion_xs, departure_ratio, T_e, (double *) dev_integral, NPHIXSPOINTS, NPHIXSNUINCREMENT);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == hipSuccess);

    double result;

    cudaStatus = hipMemcpy(&result, dev_integral, sizeof(double), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

    hipFree(dev_integral);

    return result;
}
